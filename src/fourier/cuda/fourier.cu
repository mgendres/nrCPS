#include <fourier.h>
#include "global_job_parameter.h"
#include "verbose.h"
#include "error.h"

#include "cuda_utils.h"

//---- Some file scoped variables
Fourier::Fourier(int n)
{

  const char* fname = "void Fourier::Initialize()";

  VRB.Debug(fname, "Allocating memory and creating plans for CUFFT.");

  batch_size = n;
  Cuda::Malloc((void**)&b, batch_size*GJP.Vol()*sizeof(fftComplex));

  int dims[3] = { GJP.Xsites() , GJP.Ysites(), GJP.Zsites() };
#ifdef USE_SINGLE
  hipfftPlanMany(&plan, 3, dims , NULL, 1, 0, NULL, 1, 0, HIPFFT_C2C, batch_size);
#endif
#ifdef USE_DOUBLE
  hipfftPlanMany(&plan, 3, dims , NULL, 1, 0, NULL, 1, 0, HIPFFT_Z2Z, batch_size);
#endif
#ifdef USE_LONG_DOUBLE
#error "USE_GPU not supported with USE_LONG_DOUBLE."
#endif

}

Fourier::~Fourier()
{

  const char* fname = "void Fourier::Finalize()";

  VRB.Debug(fname, "Deallocating memory and destoying plans for CUFFT.");
  Cuda::Free(b);
  hipfftDestroy(plan);

}

void Fourier::Forward()
{

  const char* fname = "void Fourier::Forward()";

#ifdef USE_SINGLE
  cufft_result = hipfftExecC2C(plan, b, b, HIPFFT_FORWARD);
#endif
#ifdef USE_DOUBLE
  cufft_result = hipfftExecZ2Z(plan, b, b, HIPFFT_FORWARD);
#endif
#ifdef USE_LONG_DOUBLE
#error "USE_GPU not supported with USE_LONG_DOUBLE."
#endif

  if (cufft_result != HIPFFT_SUCCESS) { ERR.General(fname, "Failed to CUFFT; error code: %d.", cufft_result); }
    
}

void Fourier::Backward()
{

  const char* fname = "void Fourier::Backward()";

#ifdef USE_SINGLE
  cufft_result = hipfftExecC2C(plan, b, b, HIPFFT_BACKWARD);
#endif
#ifdef USE_DOUBLE
  cufft_result = hipfftExecZ2Z(plan, b, b, HIPFFT_BACKWARD);
#endif
#ifdef USE_LONG_DOUBLE
#error "USE_GPU not supported with USE_LONG_DOUBLE."
#endif

  if (cufft_result != HIPFFT_SUCCESS) { ERR.General(fname, "Failed to CUFFT; error code: %d.", cufft_result); }

}
